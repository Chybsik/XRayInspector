
#include <hip/hip_runtime.h>
extern "C" __global__ void render(float *fb) {
    int o = threadIdx.x + blockIdx.x * blockDim.x;
    int s = threadIdx.y + blockIdx.y * blockDim.y;
	int i = o % 1280;
    if(o > 921600) return;
    fb[o] = float(i) / 1280;
}
/*
extern "C" __global__ void matSum(int *a, int *b, int *c) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
	i = i % 1280;
	j = i / 1280;
    if((i >= 1280) || (j >= 720)) return;
    int pixel_index = j*1280*3 + i*3;
    c[pixel_index + 0] = i;
    c[pixel_index + 1] = j;
    c[pixel_index + 2] = 1;
}*/
/*
__global__ void render(float *fb, int max_x, int max_y) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x*3 + i*3;
    fb[pixel_index + 0] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2;
}
*/